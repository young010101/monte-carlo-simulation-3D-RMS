#include "hip/hip_runtime.h"
#include "Simparams.h"
#include <fstream>

SimulationParameters LoadSimulationParameters(const std::string &filename)
{
    SimulationParameters params{};
    std::ifstream file(filename);
    file >> params.dt >> params.TN >> params.NPar >> params.Nbvec >>
        params.Nc >> params.res >> params.NPix1 >> params.NPix2 >> params.NPix3;
    return params;
}

thrust::host_vector<double> LoadVectorFromFile(const std::string &filename)
{
    // cy, I wish add the num of vector
    thrust::host_vector<double> vector;
    std::ifstream file(filename);
    double value;
    while (file >> value)
    {
        vector.push_back(value);
    }
    return vector;
}

Compute_params load_compute_params(const thrust::host_vector<double>& T2_real, const SimulationParameters &sim_params, const thrust::host_vector<double> &D)
{

    auto T2{T2_real};
    for (auto &a : T2)
    {
        // cy, change the real T2 to unreal but unit(?can fit to unit box) T2
        a /= sim_params.dt;
        // std::cout << a << std::endl;
    }

    thrust::host_vector<double> step(sim_params.Nc);
    for (int i = 0; i < sim_params.Nc; ++i)
    // for(auto &a: step)
    {
        // cy, no the real step size, the unit (or normalization) step size in voxel box
        // cy, dt, D, and res is realistic.
        step[i] = sqrt(6.0 * sim_params.dt * D[i]) / sim_params.res;
        std::cout << "step size=" << step[i] << std::endl;
    }

    thrust::host_vector<double> Pij(sim_params.Nc * sim_params.Nc);
    int k = 0;
    for (int i = 0; i < sim_params.Nc; ++i)
    {
        for (int j = 0; j < sim_params.Nc; ++j)
        {
            if ((i == 0) || (j == 0))
            {
                Pij[k] = 0.0;
            }
            else if (i == j)
            {
                Pij[k] = 1.0;
            }
            else
            {
                Pij[k] = std::min(1.0, sqrt(D[j] / D[i]));
            }
            std::cout << k << " permeation probability=" << Pij[k] << std::endl;
            ++k;
        }
    }

    Compute_params compute_params(T2, step, Pij);
    return compute_params;
}
// int load_pgse_params(){

// }
